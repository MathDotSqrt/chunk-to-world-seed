#include "hip/hip_runtime.h"
//MathDotSqrt

//"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v10.0\bin\nvcc.exe"  -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -o main main.cu -O3


// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __HIPCC__
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_cmath.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <hip/device_functions.h>
#endif


#include <iostream>
#include <math.h>
#include <cmath>
#include <vector>
#include <sstream>
#include <string>
#include <fstream>

//Used for sleep function
#include <chrono>
#include <thread>

#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

/*MAGIC NUMBERS*/
constexpr uint64_t MASK48 = ((1ULL << 48) - 1ULL);
constexpr uint64_t MASK32 = ((1ULL << 32) - 1ULL);
constexpr uint64_t MASK16 = ((1ULL << 16) - 1ULL);
constexpr uint64_t M1 = 25214903917ULL;
constexpr uint64_t APPEND1 = 11ULL;

constexpr uint64_t M2 = 205749139540585ULL;
constexpr uint64_t ADDEND2 = 277363943098ULL;

constexpr uint64_t M4 = 55986898099985ULL;
constexpr uint64_t APPEND4 = 49720483695876ULL;
/*MAGIC NUMBERS*/


constexpr int BLOCK_SIZE = 256;
constexpr int NUM_BLOCKS = 256
constexpr int NUM_WORKERS = NUM_BLOCKS * BLOCK_SIZE;





constexpr int CHUNK_X = 3;
constexpr int CHUNK_Z = -3;

constexpr int MAX_LINE = 1000;
