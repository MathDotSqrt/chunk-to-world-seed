//"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v10.0\bin\nvcc.exe"  -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -o main main.cu -O3

// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_cmath.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <device_functions.h>
#endif


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cmath>
#include <vector>
#include <sstream>
#include <string>
#include <fstream>

//Used for sleep function
#include <chrono>
#include <thread>

#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n",
        hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

#ifndef CHUNK_X
#define CHUNK_X 3
#endif
#define CHUNK_Z -3
#define OUTPUT_SEED_ARRAY_SIZE (1ULL << 20)
#define WORKER_COUNT (1ULL << 16)
#define MAXCHAR 1000

#define MASK48 ((1ULL << 48) - 1ULL)
#define MASK32 ((1ULL << 32) - 1ULL)
#define MASK16 ((1ULL << 16) - 1ULL)

#define M1 25214903917ULL
#define ADDEND1 11ULL

#define M2 205749139540585ULL
#define ADDEND2 277363943098ULL

#define M4 55986898099985ULL
#define ADDEND4 49720483695876ULL

__host__ __device__ int64_t nextLong(uint64_t* seed) {
    *seed = (*seed * M1 + ADDEND1) & MASK48;
    int32_t u = *seed >> 16;
    *seed = (*seed * M1 + ADDEND1) & MASK48;
    return ((uint64_t)u << 32) + (int32_t)(*seed >> 16);
}

__device__ void addSeed(uint64_t seed, uint64_t* seeds, uint64_t* seedCounter)
{
    // unsigned long long* cast is required for CUDA 9 :thonkgpu:
    uint64_t id = atomicAdd((unsigned long long*) seedCounter, 1ULL);
    seeds[id] = seed;
}

inline __host__ __device__ uint64_t makeMask(int32_t bits) {
    return (1ULL << bits) - 1;
}

__host__ int countTrailingZeroesHost(uint64_t v){
    int c = 0;
    v = (v ^ (v - 1)) >> 1;

    for(c = 0; v != 0; c++){
        v >>= 1;
    }

    return c;
}

__device__ int countTrailingZeroes(uint64_t v) {
    return __popcll((v & (-v))-1);
}

constexpr __host__ __device__ uint64_t modInverse(uint64_t x) {
    uint64_t inv = 0;
    uint64_t b = 1;
    for (int32_t i = 0; i < 16; i++) {
        inv |= (1ULL << i) * (b & 1);
        b = (b - x * (b & 1)) >> 1;
    }
    return inv;
}

__host__ __device__ uint64_t getChunkSeed(uint64_t worldSeed) {
    uint64_t seed = (worldSeed ^ M1) & MASK48;
    int64_t a = nextLong(&seed) / 2 * 2 + 1;
    int64_t b = nextLong(&seed) / 2 * 2 + 1;
    return (uint64_t)(((CHUNK_X * a + CHUNK_Z * b) ^ worldSeed) & MASK48);
}

__host__ __device__ uint64_t getPartialAddend(uint64_t partialSeed, int32_t bits) {
    uint64_t mask = makeMask(bits);
    return ((uint64_t)CHUNK_X) * (((int32_t)(((M2 * ((partialSeed ^ M1) & mask) + ADDEND2) & MASK48) >> 16)) / 2 * 2 + 1) +
           ((uint64_t)CHUNK_Z) * (((int32_t)(((M4 * ((partialSeed ^ M1) & mask) + ADDEND4) & MASK48) >> 16)) / 2 * 2 + 1);
}

__device__ void addWorldSeed(uint64_t firstAddend, int32_t multTrailingZeroes, uint64_t firstMultInv,
                                    uint64_t c, uint64_t chunkSeed, uint64_t* seeds, uint64_t* seedCounter) {
    if(countTrailingZeroes(firstAddend) < multTrailingZeroes)
        return;
    uint64_t bottom32BitsChunkseed = chunkSeed & MASK32;

    uint64_t b = (((firstMultInv * firstAddend) >> multTrailingZeroes) ^ (M1 >> 16)) & makeMask(16 - multTrailingZeroes);
    if (multTrailingZeroes != 0) {
        uint64_t smallMask = makeMask(multTrailingZeroes);
        uint64_t smallMultInverse = smallMask & firstMultInv;
        uint64_t target = (((b ^ (bottom32BitsChunkseed >> 16)) & smallMask) -
                                (getPartialAddend((b << 16) + c, 32 - multTrailingZeroes) >> 16)) & smallMask;
        b += (((target * smallMultInverse) ^ (M1 >> (32 - multTrailingZeroes))) & smallMask) << (16 - multTrailingZeroes);
    }
    uint64_t bottom32BitsSeed = (b << 16) + c;
    uint64_t target2 = (bottom32BitsSeed ^ bottom32BitsChunkseed) >> 16;
    uint64_t secondAddend = (getPartialAddend(bottom32BitsSeed, 32) >> 16);
    secondAddend &= MASK16;
    uint64_t topBits = ((((firstMultInv * (target2 - secondAddend)) >> multTrailingZeroes) ^ (M1 >> 32)) & makeMask(16 - multTrailingZeroes));

    for (; topBits < (1ULL << 16); topBits += (1ULL << (16 - multTrailingZeroes))) {
        if (getChunkSeed((topBits << 32) + bottom32BitsSeed) == chunkSeed) {
            addSeed((topBits << 32) + bottom32BitsSeed, seeds, seedCounter);
        }
    }

}

__global__ void crack(uint64_t seedInputCount, uint64_t* seedInputArray, uint64_t* seedOutputCounter, uint64_t* seedOutputArray,
                        int32_t multTrailingZeroes, uint64_t firstMultInv, int32_t xCount, int32_t zCount, int32_t totalCount) {
    uint64_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id > seedInputCount)
        return;

    uint64_t chunkSeed = seedInputArray[global_id];
    int32_t x = CHUNK_X;
    int32_t z = CHUNK_Z;

#if CHUNK_X == 0 && CHUNK_Z == 0
    addSeed(chunkSeed, seedOutputArray, seedOutputCounter);
#else
    uint64_t f = chunkSeed & MASK16;
    uint64_t c = xCount == zCount ? chunkSeed & ((1ULL << (xCount + 1)) - 1) :
                                    chunkSeed & ((1ULL << (totalCount + 1)) - 1) ^ (1 << totalCount);
    for (; c < (1ULL << 16); c += (1ULL << (totalCount + 1))) {
        uint64_t target = (c ^ f) & MASK16;
        uint64_t magic = (uint64_t)(x * ((M2 * ((c ^ M1) & MASK16) + ADDEND2) >> 16)) +
                         (uint64_t)(z * ((M4 * ((c ^ M1) & MASK16) + ADDEND4) >> 16));
        addWorldSeed(target - (magic & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#if CHUNK_X != 0
        addWorldSeed(target - ((magic + x) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_Z != 0 && CHUNK_X != CHUNK_Z
        addWorldSeed(target - ((magic + z) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X + CHUNK_Z != 0
        addWorldSeed(target - ((magic + x + z) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_X != 0 && CHUNK_X != CHUNK_Z
        addWorldSeed(target - ((magic + 2 * x) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_Z != 0 && CHUNK_X != CHUNK_Z
        addWorldSeed(target - ((magic + 2 * z) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X + CHUNK_Z != 0 && CHUNK_X * 2 + CHUNK_Z != 0
        addWorldSeed(target - ((magic + 2 * x + z) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X != CHUNK_Z && CHUNK_X + CHUNK_Z != 0 && CHUNK_X + CHUNK_Z * 2 != 0
        addWorldSeed(target - ((magic + x + 2 * z) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X + CHUNK_Z != 0
        addWorldSeed(target - ((magic + 2 * x + 2 * z) & MASK16), multTrailingZeroes, firstMultInv, c, chunkSeed, seedOutputArray, seedOutputCounter);
#endif
    }
#endif // !(CHUNK_X == 0 && CHUNK_Z == 0)
}

#ifndef OUTPUT_FILE
#define OUTPUT_FILE "data/BadWorldSeeds.txt"
#endif

#ifndef INPUT_FILE
#define INPUT_FILE "data/chunk_seeds.txt"
#endif

#undef int
int main() {
    #define int uint32_t
    setbuf(stdout, NULL);


    //file pointers for input and out
    FILE *fp;
    FILE *fp_out;

    //buffer for reading each line of the file
    char str[MAXCHAR];

    //calculating number of lines in input file
    fp = fopen(INPUT_FILE, "r");
    uint64_t totalInputSeeds = 0;
    if (!fp) {
        printf("Could not open file \n");
        return 1;
    }
    printf("Counting input size...\n");
    while (fgets(str, MAXCHAR, fp))
        totalInputSeeds++;
    fclose(fp);   //no need to close and reopen lmao





    //buffer for what
    uint64_t* buffer = (uint64_t*)malloc(WORKER_COUNT * sizeof(uint64_t));

    //number of input seeds
    int inputSeedCount = WORKER_COUNT;

    //array of input seeds
    uint64_t* inputSeeds;
    //doesnt need to be malloc managed
    CHECK_GPU_ERR(hipMallocManaged(&inputSeeds, sizeof(*inputSeeds) * (inputSeedCount)));

    //number of outputseeds
    uint64_t* outputSeedCount;
    CHECK_GPU_ERR(hipMallocManaged(&outputSeedCount, sizeof(*outputSeedCount)));
    //array of outputseeds
    uint64_t* outputSeeds;
    CHECK_GPU_ERR(hipMallocManaged(&outputSeeds, sizeof(*outputSeeds) * OUTPUT_SEED_ARRAY_SIZE));

    fp_out = fopen(OUTPUT_FILE, "w");
    fp = fopen(INPUT_FILE, "r");
    if (!fp) {
        printf("Could not open file\n");
        return 1;
    }
    //writes input seeds file into shared memory
    //only writes WORKER_COUNT ammount of seeds
    for(uint64_t i = 0; i < WORKER_COUNT; i++)
    {
        if(fgets(str, MAXCHAR, fp) != NULL)
        {
            sscanf(str, "%llu", &inputSeeds[i]);
        }
    }

    printf("init INPUT_SEED_COUNT %llu\n", WORKER_COUNT);


    printf("Beginning converting %lu seeds\n", totalInputSeeds);
    int count = 0; // Counter used for end bit
    int64_t numSearched = 0;
    int64_t totalSeeds = 0;
    clock_t lastIteration = clock();
    clock_t startTime = clock();

    uint64_t firstMultiplier = (M2 * CHUNK_X + M4 * CHUNK_Z) & MASK16;
    int32_t multTrailingZeroes = countTrailingZeroesHost(firstMultiplier);
    uint64_t firstMultInv = modInverse(firstMultiplier >> multTrailingZeroes);

    int32_t xCount = countTrailingZeroesHost(CHUNK_X);
    int32_t zCount = countTrailingZeroesHost(CHUNK_Z);
    int32_t totalCount = countTrailingZeroesHost(CHUNK_X | CHUNK_Z);
    printf("FIRST %llu\n", inputSeeds[0]);
    while (true) {
        //runs crack with WORKER_COUNT number of seeds
        std::cout << inputSeedCount << " " << multTrailingZeroes << " " << firstMultInv << " " << xCount << " " << zCount << " " << totalCount << "\n";

        crack<<<(WORKER_COUNT >> 9), (1 << 9)>>>(inputSeedCount, inputSeeds,
                                            outputSeedCount, outputSeeds,
                                            multTrailingZeroes, firstMultInv,
                                            xCount, zCount, totalCount);

        //reads more seeds while running
        bool doneFlag = false;
        count = 0;
        for(uint64_t i = 0; i < WORKER_COUNT; i++) {

            if(fgets(str, MAXCHAR, fp) != NULL) {
                sscanf(str, "%llu", &buffer[i]);
                count++;
            } else {
                doneFlag = true;
            }
        }
        printf("INPUT_SEED_COUNT %llu\n", inputSeedCount);

        CHECK_GPU_ERR(hipPeekAtLastError());
        //waits for gpu to finish before uploading new work
        CHECK_GPU_ERR(hipDeviceSynchronize());
        for(uint64_t i = 0; i < WORKER_COUNT; i++) {
            inputSeeds[i] = buffer[i];
        }

        double iterationTime = (double)(clock() - lastIteration) / CLOCKS_PER_SEC;
        double timeElapsed = (double)(clock() - startTime) / CLOCKS_PER_SEC;
        lastIteration = clock();
        numSearched += WORKER_COUNT;
        double speed = WORKER_COUNT / iterationTime / 1000.0;
        double progress = (double) numSearched / (double) totalInputSeeds * 100.0;
        double estimatedTime = (double) (totalInputSeeds - numSearched) / (double) WORKER_COUNT * iterationTime;
        char suffix = 's';
        if (estimatedTime >= 3600) {
            suffix = 'h';
            estimatedTime /= 3600;
        } else if (estimatedTime >= 60) {
            suffix = 'm';
            estimatedTime /= 60;
        }
        if (progress >= 100) {
            estimatedTime = 0;
            suffix = 's';
        }
        totalSeeds += *outputSeedCount;

        printf("Searched: %ld seeds. Found %ld matches. Uptime: %.1fs. Speed: %.2fk seeds/s. Completion: %.3f%%. ETA: %.1f%c.\n", numSearched, totalSeeds, timeElapsed, speed, progress, estimatedTime, suffix);

        for (int i = 0; i < *outputSeedCount; i++) {
            fprintf(fp_out, "%llu\n", outputSeeds[i]);
        }
        fflush(fp_out);

        *outputSeedCount = 0;
        if (doneFlag) {
            printf("DONE\n");
            break;
        }
    }


    //whyyy
    printf("COMPUTER %llu %llu\n", inputSeedCount, *outputSeedCount);
    crack<<<(WORKER_COUNT >> 9), (1 << 9)>>>(count, inputSeeds, outputSeedCount,
                                        outputSeeds, multTrailingZeroes,
                                        firstMultInv, xCount, zCount,
                                        totalCount);
    CHECK_GPU_ERR(hipDeviceSynchronize());


    //write outputseeds to file
    for (int i = 0; i < *outputSeedCount; i++) {
        fprintf(fp_out, "%llu\n", outputSeeds[i]);
    }
    fflush(fp_out);
    fclose(fp);
    fclose(fp_out);
}
